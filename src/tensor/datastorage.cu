#include "datastorage.hpp"

template <typename T>
DataStorage<T>::DataStorage(int size,Device device) {
    this->size = size;
    this->device = device;
    if (device == Device::CPU) {
        this->data = new T[size];
    } else {
        hipMalloc(&this->data,size * sizeof(T));
    }
}
template <typename T>
DataStorage<T>::DataStorage(T* data,int size,Device device){
    this->size = size;
    this->device = device;
    if (device == Device::CPU) {
        this->data = new T[size];
        memcpy(this->data,data,size * sizeof(T));
    } else {
        hipMalloc(&(this->data),size * sizeof(T));
        hipMemcpy(this->data,data,size * sizeof(T),hipMemcpyHostToDevice);
    }
}
template <typename T>
DataStorage<T>::~DataStorage() {
    if (device == Device::CPU) {
        delete[] data;
    } else {
        hipFree(data);
    }
}
template <typename T>
void DataStorage<T>::to(std::string device){
    if (device == "cpu")
        this->to(Device::CPU);
    else if (device == "cuda")
        this->to(Device::CUDA);
    else
        std::cerr<<"Invalid device"<<std::endl;
}
template <typename T>
void DataStorage<T>::to(Device device){
    switch (device){
        case Device::CPU:
            if (this->device == Device::CUDA) {
                T* temp = new T[size];
                hipMemcpy(temp,this->data,size * sizeof(T),hipMemcpyDeviceToHost);
                hipFree(this->data);
                this->data = temp;
            }
            break;
        case Device::CUDA:
            if (this->device == Device::CPU) {
                T* temp;
                hipMalloc(&temp,size * sizeof(T));
                hipMemcpy(temp,this->data,size * sizeof(T),hipMemcpyHostToDevice);
                delete[] this->data;
                this->data = temp;
            }
            break;
        default:
            std::cerr<<"Invalid device"<<std::endl;

    }
}